#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <CUDA/hiprand.h>
#include <CUDA/hiprand/hiprand_kernel.h>


#include <ExampleGame/Core/Camera.h>
#include <ExampleGame/Core/Util.h>
#include <ExampleGame/GameObject/GameObject.h>
#include <ExampleGame/GameObject/Sphere.h>
#include <ExampleGame/Material/Material.h>


#include <ExampleGame/CUDA/kernel.cuh>

#include <SFML/Graphics.hpp>

#include <algorithm>
#include <fstream>
#include <iostream>
#include <stdio.h>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}


__global__ void LaiEngine::CUDA::kernel::CreateWorld(LaiEngine::GameObject ** objects, LaiEngine::GameObject** world, LaiEngine::CUDA::Camera** camera)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		objects[0] = new LaiEngine::Sphere(glm::vec3(0, 0, -1), 0.5f, new Lambertian(glm::vec3(0.8f, 0.3f, 0.3f)));
		objects[1] = new LaiEngine::Sphere(glm::vec3(0, -100.5, -1), 100, new Lambertian(glm::vec3(0.8f, 0.8f, 0.0f)));
		objects[2] = new LaiEngine::Sphere(glm::vec3(1.0f, 0, -1.0f), 0.5f, new Metal(glm::vec3(0.8f, 0.6f, 0.2f), 0.6f));
		objects[3] = new LaiEngine::Sphere(glm::vec3(-1.0f, 0, -1.0f), 0.5f, new Metal(glm::vec3(0.8f, 0.8f, 0.8f), 0.1f));

		*world = new LaiEngine::GameObjectList(objects, 4);

		*camera = new LaiEngine::CUDA::Camera();
	}
}

__global__ void LaiEngine::CUDA::kernel::FreeWorld(LaiEngine::GameObject ** objects, LaiEngine::GameObject** world, LaiEngine::CUDA::Camera** camera)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		constexpr int numObjects = 4;

		for (int i = 0; i < numObjects; i++)
		{
			delete ((LaiEngine::Sphere *)objects[i])->pMaterial;
			delete objects[i];
		}

		delete* world;
		delete* camera;
	}
}

__global__ void LaiEngine::CUDA::kernel::InitRandState(int nx, int ny, hiprandState * randState)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= nx) || (j >= ny)) return;
	int pixel_index = j * nx + i;
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984, pixel_index, 0, &randState[pixel_index]);
}


__global__ void LaiEngine::CUDA::kernel::Render(uint8_t * outputBuffer, int max_x, int max_y, int ns, LaiEngine::CUDA::Camera** camera, LaiEngine::GameObject** world, hiprandState * randState)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i >= max_x || j >= max_y) return;


	constexpr int sizePerPixel = 4;
	int bufferIndex = j * max_x * sizePerPixel + i * sizePerPixel;

	int randStateIndex = j * max_x + i;
	hiprandState local_rand_state = randState[randStateIndex];

	glm::vec3 color;

	for (int s = 0; s < ns; s++)
	{
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
		float v = float(max_y - j + 1 + hiprand_uniform(&local_rand_state)) / float(max_y);

		LaiEngine::Ray ray = (*camera)->GetRay(u, v);

		color += GetColors(ray, world, &local_rand_state);
	}

	color /= static_cast<float>(ns);

	color.x = sqrt(color.x);
	color.y = sqrt(color.y);
	color.z = sqrt(color.z);

	uint8_t r = static_cast<uint8_t>(255.99f * color.x);
	uint8_t g = static_cast<uint8_t>(255.99f * color.y);
	uint8_t b = static_cast<uint8_t>(255.99f * color.z);


	outputBuffer[bufferIndex + 0] = r;
	outputBuffer[bufferIndex + 1] = g;
	outputBuffer[bufferIndex + 2] = b;
	outputBuffer[bufferIndex + 3] = 255;
}



__device__ glm::vec3 LaiEngine::CUDA::GetColors(const LaiEngine::Ray & ray, LaiEngine::GameObject** world, hiprandState *randState)
{
	LaiEngine::Ray currentRay = ray;

	glm::vec3 attenuation = glm::vec3(1.0f);

	for (int i = 0; i < 50; i++)
	{
		LaiEngine::Util::ShadeRec rec;

		if (world != nullptr && (*world)->Hit(currentRay, 0.001f, 1000000.0f, rec))
		{
			LaiEngine::Ray scattered;
			glm::vec3 tempAttenuation;

			if (rec.pMaterial->Scatter(currentRay, rec, tempAttenuation, scattered, randState))
			{
				attenuation *= tempAttenuation;
				currentRay = scattered;
			}
			else
			{
				return glm::vec3(0.0f);
			}
		}
		else
		{
			glm::vec3 unit_direction = glm::normalize(currentRay.Direction);

			float t = 0.5f * (unit_direction.y + 1.0f);
			glm::vec3 c = (1.0f - t) * glm::vec3(1.0f, 1.0f, 1.0f) + t * glm::vec3(0.5f, 0.7f, 1.0f);

			return attenuation * c;
		}
	}

	return glm::vec3(0.0f);
}
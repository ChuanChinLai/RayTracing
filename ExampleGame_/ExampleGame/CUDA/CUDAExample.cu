#include "hip/hip_runtime.h"
#include "CUDAExample.h"

#include <ExampleGame/CUDA/kernel.cuh>
#include <ExampleGame/Core/Camera.h>
#include <ExampleGame/Core/Util.h>
#include <ExampleGame/GameObject/GameObject.h>
#include <ExampleGame/GameObject/Sphere.h>
#include <ExampleGame/Material/Material.h>



#include <ctime>
#include <iostream>

#define checkCudaErrors(val) check_cuda1( (val), #val, __FILE__, __LINE__ )

void check_cuda1(hipError_t result, char const *const func, const char *const file, int const line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}


void LaiEngine::CUDA::CUDAExample::Init(int nx, int ny, int tx, int ty)
{
	clock_t start, stop;

	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);

	start = clock();

	InitRandState(nx, ny, tx, ty);
	InitWorld();


	stop = clock();
	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "took " << timer_seconds << " seconds.\n";
}

void LaiEngine::CUDA::CUDAExample::Update(uint8_t* outputBuffer, hiprandState* randBuffer, int nx, int ny, int ns, int tx, int ty)
{
	clock_t start, stop;

	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);

	start = clock();

	// allocate device buffer
	uint8_t* deviceBuffer;
	constexpr size_t size_rgba = 4;
	const size_t buffer_size = nx * ny * size_rgba;
	checkCudaErrors(hipMallocManaged((void **)&deviceBuffer, buffer_size));


	LaiEngine::CUDA::kernel::Render << < blocks, threads >> > (deviceBuffer, nx, ny, ns, camera, world, randState);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemcpy(outputBuffer, deviceBuffer, buffer_size, hipMemcpyDeviceToHost));
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(deviceBuffer));

	stop = clock();
	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "took " << timer_seconds << " seconds.\n";
}

void LaiEngine::CUDA::CUDAExample::Free()
{
	LaiEngine::CUDA::kernel::FreeWorld << <1, 1 >> > (gameobjects, world, camera);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(randState));

	checkCudaErrors(hipFree(gameobjects));
	checkCudaErrors(hipFree(world));
	checkCudaErrors(hipFree(camera));

	hipDeviceReset();
}

void LaiEngine::CUDA::CUDAExample::InitRandState(const int nx, const int ny, const int tx, const int ty)
{
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);

	const size_t bufferSize = nx * ny * sizeof(hiprandState);

	checkCudaErrors(hipMalloc((void **)&randState, bufferSize));
	LaiEngine::CUDA::kernel::InitRandState << <blocks, threads >> > (nx, ny, randState);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

void LaiEngine::CUDA::CUDAExample::InitWorld()
{
	constexpr int numObjects = 4;
	checkCudaErrors(hipMalloc((void **)&gameobjects, numObjects * sizeof(LaiEngine::GameObject*)));
	checkCudaErrors(hipMalloc((void **)&world, sizeof(LaiEngine::GameObject*)));
	checkCudaErrors(hipMalloc((void **)&camera, sizeof(LaiEngine::CUDA::Camera*)));

	LaiEngine::CUDA::kernel::CreateWorld << <1, 1>> > (gameobjects, world, camera);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

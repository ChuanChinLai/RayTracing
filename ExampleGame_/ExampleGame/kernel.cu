#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include ""

#include <stdio.h>

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;

	c[i] = a[i] + b[i];

}

__global__ void vector_add(double *a, double *b, double *c)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = 0; i < 10; i++)
	{
		for (int j = 0; j < 10000; j++)
		{
			c[index] = a[index] * a[index] + b[index] * b[index];
		}
	}
}


// Helper function for using CUDA to add vectors in parallel.

void addWithCuda(double *a, double *b, double *c, unsigned int size)
{
	int N = 1024 * 1024;

	double *d_a, *d_b, *d_c;

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);


	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	vector_add << < (N + (1024 - 1)) / 1024, 1024 >> > (d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);


	printf("c[%d] = %f\n", 0, c[0]);
	printf("c[%d] = %f\n", N - 1, c[N - 1]);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <CUDA/hiprand.h>
#include <CUDA/hiprand/hiprand_kernel.h>

#include <ExampleGame/Core/Utility.h>
#include <ExampleGame/GameObject/GameObject.h>
#include <ExampleGame/GameObject/Sphere.h>
#include <SFML/Graphics.hpp>

#include <fstream>
#include <iostream>
#include <stdio.h>


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) 
{
	if (result) 
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

__global__ void render_init(int max_x, int max_y, hiprandState *randState) 
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984, pixel_index, 0, &randState[pixel_index]);
}


__device__ bool hit_sphere(const glm::vec3& center, float radius, const LaiEngine::Ray& r) 
{
	glm::vec3 oc = r.Origin - center;
	float a = glm::dot(r.Direction, r.Direction);
	float b = 2.0f * dot(oc, r.Direction);
	float c = dot(oc, oc) - radius * radius;
	float discriminant = b * b - 4.0f*a*c;
	return (discriminant > 0.0f);
}



__device__ glm::vec3 color(const LaiEngine::Ray& r, LaiEngine::GameObject **world)
{
	LaiEngine::Util::ShadeRec rec; 

	if (world != nullptr && (*world)->Hit(r, 0.0f, 100000.0f, rec))
	{
		return 0.5f * glm::vec3(rec.normal.x + 1.0f, rec.normal.y + 1.0f, rec.normal.z + 1.0f);
	}


	glm::vec3 unit_direction = glm::normalize(r.Direction);
	float t = 0.5f * (unit_direction.y + 1.0f);
	return (1.0f - t) * glm::vec3(1.0, 1.0, 1.0) + t * glm::vec3(0.5, 0.7, 1.0);
}

__global__ void render(uint8_t* outputBuffer, int max_x, int max_y, LaiEngine::GameObject **world)
{

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i >= max_x || j >= max_y) return;

	float u = float(i) / float(max_x);
	float v = float(max_y - j + 1) / float(max_y);

	LaiEngine::Ray ray(glm::vec3(0.0), glm::vec3(-1, -1, -1) + u * glm::vec3(2, 0, 0) + v * glm::vec3(0, 2, 0));

	glm::vec3 c = color(ray, world);

	float r = c.x;
	float g = c.y;
	float b = c.z;

	uint8_t ir = static_cast<uint8_t>(255.99f * r);
	uint8_t ig = static_cast<uint8_t>(255.99f * g);
	uint8_t ib = static_cast<uint8_t>(255.99f * b);


	int pixel_index = j * max_x * 4 + i * 4;

	outputBuffer[pixel_index + 0] = ir;
	outputBuffer[pixel_index + 1] = ig;
	outputBuffer[pixel_index + 2] = ib;
	outputBuffer[pixel_index + 3] = 255;


	//int pixel_index = j * max_x * 4 + i * 4;

	//deviceBuffer[pixel_index + 0] = static_cast<float>(i) / max_x;
	//deviceBuffer[pixel_index + 1] = static_cast<float>(j) / max_y;
	//deviceBuffer[pixel_index + 2] = 0.2;

	//float r = deviceBuffer[pixel_index + 0];
	//float g = deviceBuffer[pixel_index + 1];
	//float b = deviceBuffer[pixel_index + 2];

	//uint8_t ir = static_cast<uint8_t>(255.99f * r);
	//uint8_t ig = static_cast<uint8_t>(255.99f * g);
	//uint8_t ib = static_cast<uint8_t>(255.99f * b);

	//outputBuffer[pixel_index + 0] = ir;
	//outputBuffer[pixel_index + 1] = ig;
	//outputBuffer[pixel_index + 2] = ib;
	//outputBuffer[pixel_index + 3] = 255;
}

__global__ void create_world(LaiEngine::GameObject **d_list, LaiEngine::GameObject **d_world)
{
	if (threadIdx.x == 0 && blockIdx.x == 0) 
	{
		*(d_list) = new LaiEngine::Sphere(glm::vec3(0, 0, -1), 0.5, nullptr);
		*(d_list + 1) = new LaiEngine::Sphere(glm::vec3(0, -100.5, -1), 100, nullptr);
		*d_world = new LaiEngine::GameObjectList(d_list, 2);
	}
}


__global__ void free_world(LaiEngine::GameObject **d_list, LaiEngine::GameObject **d_world)
{
	delete *(d_list);
	delete *(d_list + 1);
	delete *d_world;
}


// Helper function for using CUDA to add vectors in parallel.


void RenderWithCuda(uint8_t* outputBuffer, const size_t buffer_size, int nx, int ny, int tx, int ty)
{
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);


	// allocate random state
	hiprandState *d_randState;
	checkCudaErrors(hipMalloc((void **)&d_randState, nx * ny * sizeof(hiprandState)));
	render_init << <blocks, threads >> > (nx, ny, d_randState);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());


	// allocate device buffer
	uint8_t* deviceBuffer;
	checkCudaErrors(hipMallocManaged((void **)&deviceBuffer, buffer_size));

	// make our world of hitables
	LaiEngine::GameObject **d_List;
	checkCudaErrors(hipMalloc((void **)&d_List, 2 * sizeof(LaiEngine::GameObject* )));
	LaiEngine::GameObject **d_World;
	checkCudaErrors(hipMalloc((void **)&d_World, sizeof(LaiEngine::GameObject* )));

	create_world << < 1, 1 >> > (d_List, d_World);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());


	clock_t start, stop;
	start = clock();
	render<<< blocks, threads >>>(deviceBuffer, nx, ny, d_World);

	stop = clock();
	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "took " << timer_seconds << " seconds.\n";


	checkCudaErrors(hipMemcpy(outputBuffer, deviceBuffer, buffer_size, hipMemcpyDeviceToHost));

	checkCudaErrors(hipDeviceSynchronize());
	free_world << <1, 1 >> > (d_List, d_World);

	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipFree(d_randState));
	checkCudaErrors(hipFree(d_List));
	checkCudaErrors(hipFree(d_World));
	checkCudaErrors(hipFree(deviceBuffer));

	hipDeviceReset();
}